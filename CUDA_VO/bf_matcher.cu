#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>

#include "ORB_CPU.h"
#include <hip/hip_runtime.h>

__global__ void BfMatchKernel(const DescType* desc1, const DescType* desc2, cv::DMatch* matches, int d_max, int desc1Size, int desc2Size) {
  int i1 = blockIdx.x * blockDim.x + threadIdx.x;
  


}

void BfMatch_CUDA(const vector<DescType> &desc1, const vector<DescType> &desc2, vector<cv::DMatch> &matches) {
  const int d_max = 40;
  int desc1Size = desc1.size();
  int desc2Size = desc2.size();

  // Allocate device memory
  DescType* d_desc1;
  DescType* d_desc2;
  cv::DMatch* d_matches;

  hipMalloc((void**)&d_desc1, desc1Size * sizeof(DescType));
  hipMalloc((void**)&d_desc2, desc2Size * sizeof(DescType));
  hipMalloc((void**)&d_matches, desc1Size * sizeof(cv::DMatch));

  // Copy data from host to device
  hipMemcpy(d_desc1, desc1.data(), desc1Size * sizeof(DescType), hipMemcpyHostToDevice);
  hipMemcpy(d_desc2, desc2.data(), desc2Size * sizeof(DescType), hipMemcpyHostToDevice);

  // Launch the CUDA kernel
  int threadsPerBlock = 256;
  int blocksPerGrid = (desc1Size + threadsPerBlock - 1) / threadsPerBlock;

  BfMatchKernel<<<blocksPerGrid, threadsPerBlock>>>(d_desc1, d_desc2, d_matches, d_max, desc1Size, desc2Size);

  // Copy results back to the host
  hipMemcpy(matches.data(), d_matches, desc1Size * sizeof(cv::DMatch), hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_desc1);
  hipFree(d_desc2);
  hipFree(d_matches);
}
int main()
{

	return 0;
}